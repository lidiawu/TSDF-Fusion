
#include <hip/hip_runtime.h>
/* This file is part of the Marching Cubes GPU based algorithm based on 
 * Paul Bourke's tabulation approach to marching cubes
 * http://paulbourke.net/geometry/polygonise/
 *
 *
 * We model cubes with 8 vertices labelled as below
 *
 *
 *            4--------(4)---------5
 *           /|                   /|
 *          / |                  / |
 *         /  |                 /  |
 *       (7)  |               (5)  |
 *       /    |               /    |
 *      /    (8)             /    (9)
 *     /      |             /      |
 *    7---------(6)--------6       |
      |       |            |       |
 *    |       0------(0)---|-------1
 *    |      /             |      /
 *   (11)   /             (10)   /
 *    |    /               |    /
 *    |  (3)               |  (1)
 *    |  /                 |  /
 *    | /                  | /
 *    |/                   |/
 *    3---------(2)--------2
 *
 * where X axis is horizontal, +ve to right
 *       Y axis is vertical, +ve upwards
 *       Z axis is into page, +ve towards back
 *
 * 0: ( x,   y,   z+1 )  4: ( x,   y+1,   z+1 )
 * 1: ( x+1, y,   z+1 )  5: ( x+1, y+1,   z+1 )
 * 2: ( x+1, y,   z   )  6: ( x+1, y+1,   z   )
 * 3: ( x,   y,   z   )  7: ( x,   y+1,   z   )
 *
 * There are 12 edges, 0 - 11 where each edge connectes two vertices as follows:
 *
 * 0: 0, 1       1: 1, 2       2: 2, 3       3:  3, 0
 * 4: 4, 5       5: 5, 6       6: 6, 7       7:  7, 4
 * 8: 0, 4       9: 1, 5      10: 2, 6      11:  3, 7
 */

 // NB Below, these are ordered from lower to higher value

typedef unsigned char      uint8_t;
typedef unsigned short     uint16_t;

 __constant__
uint16_t EDGE_VERTICES[12][2] = {
    { 0, 1 }, { 2, 1 }, { 3, 2 }, { 3, 0 },
    { 4, 5 }, { 6, 5 }, { 7, 6 }, { 7, 4 },
    { 0, 4 }, { 1, 5 }, { 2, 6 }, { 3, 7 } 
 };

/*
 * This file describes the relationship between the vertices under the surface
 * and the edges which are therefore impacted
 * There are 256 distinct entries
 */
__constant__
uint16_t EDGE_TABLE[256]={
	0x0  , 0x109, 0x203, 0x30a, 0x406, 0x50f, 0x605, 0x70c,
	0x80c, 0x905, 0xa0f, 0xb06, 0xc0a, 0xd03, 0xe09, 0xf00,
	0x190, 0x99 , 0x393, 0x29a, 0x596, 0x49f, 0x795, 0x69c,
	0x99c, 0x895, 0xb9f, 0xa96, 0xd9a, 0xc93, 0xf99, 0xe90,
	0x230, 0x339, 0x33 , 0x13a, 0x636, 0x73f, 0x435, 0x53c,
	0xa3c, 0xb35, 0x83f, 0x936, 0xe3a, 0xf33, 0xc39, 0xd30,
	0x3a0, 0x2a9, 0x1a3, 0xaa , 0x7a6, 0x6af, 0x5a5, 0x4ac,
	0xbac, 0xaa5, 0x9af, 0x8a6, 0xfaa, 0xea3, 0xda9, 0xca0,
	0x460, 0x569, 0x663, 0x76a, 0x66 , 0x16f, 0x265, 0x36c,
	0xc6c, 0xd65, 0xe6f, 0xf66, 0x86a, 0x963, 0xa69, 0xb60,
	0x5f0, 0x4f9, 0x7f3, 0x6fa, 0x1f6, 0xff , 0x3f5, 0x2fc,
	0xdfc, 0xcf5, 0xfff, 0xef6, 0x9fa, 0x8f3, 0xbf9, 0xaf0,
	0x650, 0x759, 0x453, 0x55a, 0x256, 0x35f, 0x55 , 0x15c,
	0xe5c, 0xf55, 0xc5f, 0xd56, 0xa5a, 0xb53, 0x859, 0x950,
	0x7c0, 0x6c9, 0x5c3, 0x4ca, 0x3c6, 0x2cf, 0x1c5, 0xcc ,
	0xfcc, 0xec5, 0xdcf, 0xcc6, 0xbca, 0xac3, 0x9c9, 0x8c0,
	0x8c0, 0x9c9, 0xac3, 0xbca, 0xcc6, 0xdcf, 0xec5, 0xfcc,
	0xcc , 0x1c5, 0x2cf, 0x3c6, 0x4ca, 0x5c3, 0x6c9, 0x7c0,
	0x950, 0x859, 0xb53, 0xa5a, 0xd56, 0xc5f, 0xf55, 0xe5c,
	0x15c, 0x55 , 0x35f, 0x256, 0x55a, 0x453, 0x759, 0x650,
	0xaf0, 0xbf9, 0x8f3, 0x9fa, 0xef6, 0xfff, 0xcf5, 0xdfc,
	0x2fc, 0x3f5, 0xff , 0x1f6, 0x6fa, 0x7f3, 0x4f9, 0x5f0,
	0xb60, 0xa69, 0x963, 0x86a, 0xf66, 0xe6f, 0xd65, 0xc6c,
	0x36c, 0x265, 0x16f, 0x66 , 0x76a, 0x663, 0x569, 0x460,
	0xca0, 0xda9, 0xea3, 0xfaa, 0x8a6, 0x9af, 0xaa5, 0xbac,
	0x4ac, 0x5a5, 0x6af, 0x7a6, 0xaa , 0x1a3, 0x2a9, 0x3a0,
	0xd30, 0xc39, 0xf33, 0xe3a, 0x936, 0x83f, 0xb35, 0xa3c,
	0x53c, 0x435, 0x73f, 0x636, 0x13a, 0x33 , 0x339, 0x230,
	0xe90, 0xf99, 0xc93, 0xd9a, 0xa96, 0xb9f, 0x895, 0x99c,
	0x69c, 0x795, 0x49f, 0x596, 0x29a, 0x393, 0x99 , 0x190,
	0xf00, 0xe09, 0xd03, 0xc0a, 0xb06, 0xa0f, 0x905, 0x80c,
	0x70c, 0x605, 0x50f, 0x406, 0x30a, 0x203, 0x109, 0x0 
};