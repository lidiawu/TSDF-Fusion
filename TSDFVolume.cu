#include "hip/hip_runtime.h"
#include "TSDFVolume.h"
#include <iostream>
#include <string>

using namespace std;

__host__
TSDFVolume::TSDFVolume(int x, int y, int z, float ox, float oy, float oz, float size){
		m_size.x = x;
		m_size.y = y;
		m_size.z = z;

		origin.x = ox;
		origin.y = oy;
		origin.z = oz;

		voxel_size = size;
		trunc_margin = voxel_size * 5;

		hipError_t err;
		size_t data_size = x * y * z * sizeof( float );

        err = hipMalloc( &m_distances, data_size );
		if(err != hipSuccess)
			cout <<  "Couldn't allocate space for distance data for TSDF" << endl;
		float * voxel_grid_TSDF = new float[x * y * z];
		for(int i = 0; i< x*y*z;i++)
			voxel_grid_TSDF[i] = 1.0f;
		hipMemcpy(m_distances, voxel_grid_TSDF, data_size, hipMemcpyHostToDevice);

        err = hipMalloc( &m_weights, data_size );
		if (err != hipSuccess)
			cout << "Couldn't allocate space for weight data for TSDF" << endl;
		hipMemset(m_weights,0,data_size);
       
		err = hipMalloc(&m_deform, x * y * z * sizeof( float3 ));
		if(err != hipSuccess)
			cout << "Couldn't allocate space for deformation data for TSDF" << endl;
		hipMemset(m_deform, 0,x * y * z * sizeof( float3 ));
	}

TSDFVolume::~TSDFVolume() {
    std::cout << "Destroying TSDFVolume" << std::endl;
    deallocate( );
}


/**
 * Deallocate storage for this TSDF
 */
void TSDFVolume::deallocate( ) {
    // Remove existing data
    if ( m_distances ) {
        hipFree( m_distances ) ;
        m_distances = 0;
    }
    if ( m_weights ) {
        hipFree( m_weights );
        m_weights = 0;
    }
    if ( m_deform ) {
        hipFree( m_deform );
        m_deform = 0;
    }
}

__global__
void Integrate_kernal(float * cam_K, float * cam2base, float * depth_im,
               dim3 size, float3 origin, float voxel_size, float trunc_margin,
               float * voxel_grid_TSDF, float * voxel_grid_weight) {

  int pt_grid_z = blockIdx.x;
  int pt_grid_y = threadIdx.x;

  for (int pt_grid_x = 0; pt_grid_x < size.x; ++pt_grid_x) {

    // Convert voxel center from grid coordinates to base frame camera coordinates
    float pt_base_x = origin.x + pt_grid_x * voxel_size;
    float pt_base_y = origin.y + pt_grid_y * voxel_size;
    float pt_base_z = origin.z + pt_grid_z * voxel_size;

    // Convert from base frame camera coordinates to current frame camera coordinates
    float tmp_pt[3] = {0};
    tmp_pt[0] = pt_base_x - cam2base[0 * 4 + 3];
    tmp_pt[1] = pt_base_y - cam2base[1 * 4 + 3];
    tmp_pt[2] = pt_base_z - cam2base[2 * 4 + 3];
    float pt_cam_x = cam2base[0 * 4 + 0] * tmp_pt[0] + cam2base[1 * 4 + 0] * tmp_pt[1] + cam2base[2 * 4 + 0] * tmp_pt[2];
    float pt_cam_y = cam2base[0 * 4 + 1] * tmp_pt[0] + cam2base[1 * 4 + 1] * tmp_pt[1] + cam2base[2 * 4 + 1] * tmp_pt[2];
    float pt_cam_z = cam2base[0 * 4 + 2] * tmp_pt[0] + cam2base[1 * 4 + 2] * tmp_pt[1] + cam2base[2 * 4 + 2] * tmp_pt[2];

    if (pt_cam_z <= 0)
      continue;

    int pt_pix_x = roundf(cam_K[0 * 3 + 0] * (pt_cam_x / pt_cam_z) + cam_K[0 * 3 + 2]);
    int pt_pix_y = roundf(cam_K[1 * 3 + 1] * (pt_cam_y / pt_cam_z) + cam_K[1 * 3 + 2]);
    if (pt_pix_x < 0 || pt_pix_x >= 640 || pt_pix_y < 0 || pt_pix_y >= 480)
      continue;

    float depth_val = depth_im[pt_pix_y * 640 + pt_pix_x];

    if (depth_val <= 0 || depth_val > 6)
      continue;

    float diff = depth_val - pt_cam_z;

    if (diff <= -trunc_margin)
      continue;

    // Integrate
    int volume_idx = pt_grid_z * size.y * size.x + pt_grid_y * size.x + pt_grid_x;
    float dist = fmin(1.0f, diff / trunc_margin);
    float weight_old = voxel_grid_weight[volume_idx];
    float weight_new = weight_old + 1.0f;
    voxel_grid_weight[volume_idx] = weight_new;
    voxel_grid_TSDF[volume_idx] = (voxel_grid_TSDF[volume_idx] * weight_old + dist) / weight_new;
  }
}

__host__
void TSDFVolume::Integrate(float* depth_map,float* cam_K, float* cam2base){
	 float * gpu_cam_K;
     float * gpu_cam2base;
	 float * gpu_depth_im;

	 hipMalloc(&gpu_depth_im, 480 * 640 * sizeof(float));
	 hipMemcpy(gpu_depth_im, depth_map, 480 * 640 * sizeof(float), hipMemcpyHostToDevice);

	 hipMalloc(&gpu_cam_K, 3 * 3 * sizeof(float));
	 hipMemcpy(gpu_cam_K, cam_K, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);

	 hipMalloc(&gpu_cam2base, 4 * 4 * sizeof(float));
	 hipMemcpy(gpu_cam2base, cam2base, 4 * 4 * sizeof(float), hipMemcpyHostToDevice);

	 Integrate_kernal<<< m_size.z, m_size.y >>>(gpu_cam_K, gpu_cam2base, gpu_depth_im, m_size, origin, voxel_size, trunc_margin,m_distances, m_weights);
}