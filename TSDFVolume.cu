#include "hip/hip_runtime.h"
#include "TSDFVolume.h"
#include <iostream>

using namespace std;

__host__
void TSDFVolume::TSDFVolume(){int x, int y, int z, float ox, float oy, float oz, float size}{
		dim_x = x;
		dim_y = y;
		dim_z = z;

		origin_x = ox;
		origin_y = oy;
		origin_z = oz;

		voxel_size = size;

		hipError_t err;
		size_t data_size = dim_x * dim_y * dim_z * sizeof( float );

        err = hipMalloc( &m_distances, data_size );
		if(err != hipSuccess)
			cout <<  "Couldn't allocate space for distance data for TSDF" << endl;

        err = hipMalloc( &m_weights, data_size );
		if (err != hipSuccess)
			cout << "Couldn't allocate space for weight data for TSDF" << endl;

        err = hipMalloc( &m_colours,  dim_x * dim_y * dim_z * sizeof( uchar3 ) );
		if(err != hipSuccess)
			cout << "Couldn't allocate space for colour data for TSDF" << endl;
       
	}