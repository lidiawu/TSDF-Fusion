#include "hip/hip_runtime.h"
#include "TSDFVolume.h"
#include <iostream>
#include <string>
#include "cuda_utils.h"

using namespace std;

__device__
float3 bilinear( float3 c00, float3 c01, float3 c10, float3 c11){
    float3  a = c00 * 0.5 + c10 * 0.5; 
    float3  b = c01 * 0.5 + c11 * 0.5; 
    return (a * 0.5 + b * 0.5); 
}

__device__
float3 trilinear(float3 c000, float3 c001, float3 c010, float3 c011, float3 c100, float3 c101, float3 c110, float3 c111){
	float3 c0, c1;
	c0 = bilinear(c000, c100, c010, c110);
	c1 = bilinear(c001, c101, c011, c111);
	return (c0 * 0.5 + c1 * 0.5);
}


__global__
void initialize_grid(float3 * grid, dim3 grid_size, float voxel_size, float3 grid_origin ){
	int vy = threadIdx.x;
	int vz = blockIdx.x;

	// If this thread is in range
    if ( vy < grid_size.y+1 && vz < grid_size.z+1 ) {

        // The next (x_size) elements from here are the x coords
        size_t base_grid_index = (grid_size.x+1) * (grid_size.y+1) * vz + (grid_size.x+1) * vy;

        size_t grid_index = base_grid_index;
        for ( int vx = 0; vx < grid_size.x+1; vx++ ) {
            grid[grid_index].x = (float)vx * voxel_size + grid_origin.x;
            grid[grid_index].y = (float)vy * voxel_size + grid_origin.y;
            grid[grid_index].z = (float)vz * voxel_size + grid_origin.z;

            grid_index++;
        }
    }

}

__global__
void deformation( float3* grid, float3 * deformation, dim3 grid_size ) {

    // Extract the voxel Y and Z coordinates we then iterate over X
    int vy = threadIdx.x;
    int vz = blockIdx.x;
	float3 c000, c001,c010,c011,c100,c101,c110,c111;
	size_t layer_size =  (grid_size.x + 1) * (grid_size.y + 1);
    // If this thread is in range
    if ( vy < grid_size.y && vz < grid_size.z ) {

        // The next (x_size) elements from here are the x coords
		size_t base_grid_index = (grid_size.x+1) * (grid_size.y+1) * vz + (grid_size.x+1) * vy;
        size_t base_voxel_index =  ((grid_size.x * grid_size.y) * vz ) + (grid_size.x * vy);

        size_t voxel_index = base_voxel_index;
		size_t grid_index = base_grid_index;
        for ( int vx = 0; vx < grid_size.x; vx++ ) {
			c000 = grid[grid_index + grid_size.x +1];
			c001 = grid[grid_index];
			c010 = grid[grid_index + grid_size.x +1 + layer_size];
			c011 = grid[grid_index + layer_size];
			c100 = grid[grid_index + grid_size.x +2];
			c101 = grid[grid_index + 1];
			c110 = grid[grid_index + grid_size.x + 2 + layer_size];
			c111 = grid[grid_index + layer_size + 1];
            
			deformation[voxel_index] = trilinear(c000, c001,c010,c011,c100,c101,c110,c111);


            voxel_index++;
			grid_index++;
        }
    }
}

__host__
TSDFVolume::TSDFVolume(int x, int y, int z, float ox, float oy, float oz, float size){
		m_size.x = x;
		m_size.y = y;
		m_size.z = z;

		origin.x = ox;
		origin.y = oy;
		origin.z = oz;

		voxel_size = size;
		trunc_margin = voxel_size * 5;

		hipError_t err;
		size_t data_size = x * y * z * sizeof( float );

        err = hipMalloc( &m_distances, data_size );
		if(err != hipSuccess)
			cout <<  "Couldn't allocate space for distance data for TSDF" << endl;
		float * voxel_grid_TSDF = new float[x * y * z];
		for(int i = 0; i< x*y*z;i++)
			voxel_grid_TSDF[i] = 1.0f;
		hipMemcpy(m_distances, voxel_grid_TSDF, data_size, hipMemcpyHostToDevice);

        err = hipMalloc( &m_weights, data_size );
		if (err != hipSuccess)
			cout << "Couldn't allocate space for weight data for TSDF" << endl;
		hipMemset(m_weights,0,data_size);

		err = hipMalloc(&grid_coord,(x+1) * (y+1) * (z+1) * sizeof(float3));
		if(err != hipSuccess)
			cout << "Couldn't allocate space for deformation data for TSDF" << endl;
		initialize_grid<<< 501, 501 >>>(grid_coord, m_size, voxel_size, origin);
		hipDeviceSynchronize( );
       
		err = hipMalloc(&m_deform, x * y * z * sizeof( float3 ));
		if(err != hipSuccess)
			cout << "Couldn't allocate space for deformation data for TSDF" << endl;
		deformation<<< 500, 500 >>>(grid_coord, m_deform, m_size);
		hipDeviceSynchronize( );

		
	}

TSDFVolume::~TSDFVolume() {
    std::cout << "Destroying TSDFVolume" << std::endl;
    deallocate( );
}


/**
 * Deallocate storage for this TSDF
 */
void TSDFVolume::deallocate( ) {
    // Remove existing data
    if ( m_distances ) {
        hipFree( m_distances ) ;
        m_distances = 0;
    }
    if ( m_weights ) {
        hipFree( m_weights );
        m_weights = 0;
    }
    if ( m_deform ) {
        hipFree( m_deform );
        m_deform = 0;
    }
}


__global__
void Integrate_kernal(float * cam_K, float * cam2base, float * depth_im,
               dim3 size, float3 origin, float voxel_size, float trunc_margin,
               float * voxel_grid_TSDF, float * voxel_grid_weight, float3* deformation) {

	int volume_idx = blockIdx.x * 512 + threadIdx.x;
	if (volume_idx < 500 * 500 * 500){
		// Convert voxel center from grid coordinates to base frame camera coordinates
		float pt_base_x = deformation[volume_idx].x;
		float pt_base_y = deformation[volume_idx].y;
		float pt_base_z = deformation[volume_idx].z;

		// Convert from base frame camera coordinates to current frame camera coordinates
		float tmp_pt[3] = {0};
		tmp_pt[0] = pt_base_x - cam2base[0 * 4 + 3];
		tmp_pt[1] = pt_base_y - cam2base[1 * 4 + 3];
		tmp_pt[2] = pt_base_z - cam2base[2 * 4 + 3];
		float pt_cam_x = cam2base[0 * 4 + 0] * tmp_pt[0] + cam2base[1 * 4 + 0] * tmp_pt[1] + cam2base[2 * 4 + 0] * tmp_pt[2];
		float pt_cam_y = cam2base[0 * 4 + 1] * tmp_pt[0] + cam2base[1 * 4 + 1] * tmp_pt[1] + cam2base[2 * 4 + 1] * tmp_pt[2];
		float pt_cam_z = cam2base[0 * 4 + 2] * tmp_pt[0] + cam2base[1 * 4 + 2] * tmp_pt[1] + cam2base[2 * 4 + 2] * tmp_pt[2];

		if (pt_cam_z <= 0)
			return;

		int pt_pix_x = roundf(cam_K[0 * 3 + 0] * (pt_cam_x / pt_cam_z) + cam_K[0 * 3 + 2]);
		int pt_pix_y = roundf(cam_K[1 * 3 + 1] * (pt_cam_y / pt_cam_z) + cam_K[1 * 3 + 2]);
		if (pt_pix_x < 0 || pt_pix_x >= 640 || pt_pix_y < 0 || pt_pix_y >= 480)
			return;

		float depth_val = depth_im[pt_pix_y * 640 + pt_pix_x];

		if (depth_val <= 0 || depth_val > 6)
			return;

		float diff = depth_val - pt_cam_z;

		if (diff <= -trunc_margin)
			return;

		// Integrate

		float dist = fmin(1.0f, diff / trunc_margin);
		float weight_old = voxel_grid_weight[volume_idx];
		float weight_new = weight_old + 1.0f;
		voxel_grid_weight[volume_idx] = weight_new;
		voxel_grid_TSDF[volume_idx] = (voxel_grid_TSDF[volume_idx] * weight_old + dist) / weight_new;
	}
}

__host__
void TSDFVolume::Integrate(float* depth_map,float* cam_K, float* cam2base){
	 float * gpu_cam_K;
     float * gpu_cam2base;
	 float * gpu_depth_im;

	 hipMalloc(&gpu_depth_im, 480 * 640 * sizeof(float));
	 hipMemcpy(gpu_depth_im, depth_map, 480 * 640 * sizeof(float), hipMemcpyHostToDevice);

	 hipMalloc(&gpu_cam_K, 3 * 3 * sizeof(float));
	 hipMemcpy(gpu_cam_K, cam_K, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);

	 hipMalloc(&gpu_cam2base, 4 * 4 * sizeof(float));
	 hipMemcpy(gpu_cam2base, cam2base, 4 * 4 * sizeof(float), hipMemcpyHostToDevice);

	 int blocknum = ceil(500 * 500 * 500 / 512.0);
	 Integrate_kernal<<< blocknum,512 >>>(gpu_cam_K, gpu_cam2base, gpu_depth_im, m_size, origin, voxel_size, trunc_margin,m_distances, m_weights, m_deform);
}
 

