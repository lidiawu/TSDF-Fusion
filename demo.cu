#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <iomanip>
#include <sstream>
#include <string>
#include "utils.hpp"
#include "TSDFVolume.h"
#include "MarchingCubes.h"



using namespace cv;
using namespace std;


// Loads a binary file with depth data and generates a TSDF voxel volume (5m x 5m x 5m at 1cm resolution)
// Volume is aligned with respect to the camera coordinates of the first frame (a.k.a. base frame)
int main(int argc, char * argv[]) {

  // Location of camera intrinsic file
  std::string cam_K_file = "E:\\GrUVi\\wuqiw\\tsdf-mc\\data\\camera-intrinsics.txt";

  // Location of folder containing RGB-D frames and camera pose files
  std::string data_path = "E:\\GrUVi\\wuqiw\\tsdf-mc\\data\\rgbd-frames";
  int base_frame_idx = 150;
  int first_frame_idx = 150;
  float num_frames = 50;

  float cam_K[3 * 3];
  float base2world[4 * 4];
  float cam2base[4 * 4];
  float cam2world[4 * 4];
  int im_width = 640;
  int im_height = 480;
  float* depth_im = new float[480 * 640];

  // Voxel grid parameters (change these to change voxel grid resolution, etc.)
  TSDFVolume volume(500,500,500,-1.5f,-1.5f, 0.5f, 0.006f);
  

  
  // Read camera intrinsics
  std::vector<float> cam_K_vec = LoadMatrixFromFile(cam_K_file, 3, 3);
  std::copy(cam_K_vec.begin(), cam_K_vec.end(), cam_K);

  // Read base frame camera pose
  std::ostringstream base_frame_prefix;
  base_frame_prefix << std::setw(6) << std::setfill('0') << base_frame_idx;
  std::string base2world_file = data_path + "\\frame-" + base_frame_prefix.str() + ".pose.txt";
  std::vector<float> base2world_vec = LoadMatrixFromFile(base2world_file, 4, 4);
  std::copy(base2world_vec.begin(), base2world_vec.end(), base2world);

  // Invert base frame camera pose to get world-to-base frame transform 
  float base2world_inv[16] = {0};
  invert_matrix(base2world, base2world_inv);


  // Loop through each depth frame and integrate TSDF voxel grid
  for (int frame_idx = first_frame_idx; frame_idx < first_frame_idx + (int)num_frames; ++frame_idx) {

    std::ostringstream curr_frame_prefix;
    curr_frame_prefix << std::setw(6) << std::setfill('0') << frame_idx;

    // // Read current frame depth
    std::string depth_im_file = data_path + "\\frame-" + curr_frame_prefix.str() + ".depth.png";
    ReadDepth(depth_im_file, im_height, im_width, depth_im);

    // Read base frame camera pose
    std::string cam2world_file = data_path + "\\frame-" + curr_frame_prefix.str() + ".pose.txt";
    std::vector<float> cam2world_vec = LoadMatrixFromFile(cam2world_file, 4, 4);
    std::copy(cam2world_vec.begin(), cam2world_vec.end(), cam2world);

    // Compute relative camera pose (camera-to-base frame)
    multiply_matrix(base2world_inv, cam2world, cam2base);

  

   std::cout << "Fusing: " << depth_im_file << std::endl;

	volume.Integrate(depth_im,cam_K,cam2base);

  }

  // Compute surface points from TSDF voxel grid and save to point cloud .ply file
  std::cout << "Saving surface point cloud (tsdf.ply)..." << std::endl;
  vector<float3> vertices ;
  vector<int3> triangles;
  extract_surface(volume, vertices, triangles);
  write_to_ply("tsdf_test.ply",vertices,triangles);

  
  delete depth_im;
  return 0;
}


